#include "hip/hip_runtime.h"
#include "nvmatrix/include/nvmatrix_kernels.cuh"
#include "nvmatrix/include/nvmatrix_operators.cuh"
#include <THC.h>
#include <algorithm>
#include "hip/hip_runtime_api.h"


__global__ void add(float* output, float* bias, int n)
{
  int ti = blockIdx.x*blockDim.x + threadIdx.x;

  if(ti >= n)
    return;

  output[ti] = bias[0];
}

extern "C" {

  void addLinearBias(THCudaTensor* output, THCudaTensor* bias)
  {
    int n = output->size[0];
    add<<< DIVUP(n,128), 128 >>> (THCudaTensor_data(output), THCudaTensor_data(bias), n);
  }
  
  void addBias(THCudaTensor* output, THCudaTensor* bias) {
    int width = output->size[1];
    int height = output->size[0];
    float *odata = THCudaTensor_data(output);
    float *bdata = THCudaTensor_data(bias);
    dim3 threads(ADD_VEC_THREADS_X, ADD_VEC_THREADS_Y);
    dim3 blocks(std::min(512, DIVUP(width, ADD_VEC_THREADS_X)),
                std::min(NUM_BLOCKS_MAX, DIVUP(height, ADD_VEC_THREADS_Y)));
    kColVectorOp<NVMatrixBinaryOps::Add>
      <<<blocks, threads>>>(odata, bdata, odata, width, height, 
                            output->stride[0], output->stride[0], 
                            NVMatrixBinaryOps::Add());
    getLastCudaError("Kernel execution failed");
  }
  
  void gradBias(THCudaTensor* gradOutput, THCudaTensor* gradBias, float scale) {
    dim3 threads(AWR_NUM_THREADS);
    dim3 blocks(1, gradOutput->size[0]);
    kAggRows_wholerow_nosync<<<blocks, threads>>>(THCudaTensor_data(gradOutput), THCudaTensor_data(gradBias), gradOutput->size[1], gradOutput->size[0], NVMatrixAggs::Sum(), NVMatrixOps::Identity(), NVMatrixBinaryOps::SecondScaled(scale));
  }
}
